#include "hip/hip_runtime.h"
#include "errors.h"
#include "graph.cu"


#define BLOCKS 128
#define THREADS 1024


__global__ void brandesCudaSingleBlock(Graph *graph, double *centrality, double *delta, int *d, int *sigma) {
    if (threadIdx.x >= max(graph->numVertices, graph->vmapSize)) {
        return;
    }

    __shared__ int s, l;
    __shared__ bool cont;
    int w, v;
    double sum;

    if (threadIdx.x == 0) {
        s = -1;
    }
    __syncthreads();  // Sync for s

    while (s < graph->numVertices - 1) {
        if (threadIdx.x == 0) {
            ++s;
            l = 0;
            cont = true;
        }
        __syncthreads();  // Sync for s

        // Init arrays in parallel
        for (v = (int) threadIdx.x; v < graph->numVertices; v += (int) blockDim.x) {
            if (v == s) {
                d[s] = 0;
                sigma[s] = 1;
            }
            else {
                d[v] = -1;
                sigma[v] = 0;
            }
        }
        __syncthreads();  // Sync for initial values

        // Forward pass
        while (cont) {
            cont = false;

            // Forward step in parallel
            for (int i = (int) threadIdx.x; i < graph->vmapSize; i += (int) blockDim.x) {
                w = graph->vmap[i];

                if (d[w] == l) {
                    for (int j = (int) graph->vptrs[i]; j < graph->vptrs[i + 1]; ++j) {
                        v = graph->adjs[j];

                        if (d[v] == -1) {
                            d[v] = l + 1;  // d[w] + 1
                            cont = true;
                        }

                        if (d[v] == l + 1) {  // (d[v] == d[w] + 1)
                            atomicAdd(&sigma[v], sigma[w]);
                        }
                    }
                }
            }
            __syncthreads();  // Sync computations

            if (threadIdx.x == 0) {
                ++l;
            }
            __syncthreads();  // Sync for l and arrays
        }

        for (v = (int) threadIdx.x; v < graph->numVertices; v += (int) blockDim.x) {
            delta[v] = (sigma[v] != 0) ? (1 / (double) sigma[v]) : 0.0;
        }

        // Backward pass
        while (l > 1) {
            __syncthreads();  // Sync for l
            if (threadIdx.x == 0) {
                --l;
            }
            __syncthreads();  // Sync for l

            // Backward step in parallel
            for (int i = (int) threadIdx.x; i < graph->vmapSize; i += (int) blockDim.x) {
                w = graph->vmap[i];

                if (d[w] == l) {
                    sum = 0;

                    for (int j = (int) graph->vptrs[i]; j < graph->vptrs[i + 1]; ++j) {
                        v = graph->adjs[j];

                        if (d[v] == l + 1) {  // (d[v] == d[w] + 1)
                            sum += delta[v];
                        }
                    }
                    atomicAdd(&delta[w], sum);

                }

            }
            __syncthreads();
        }

        for (v = (int) threadIdx.x; v < graph->numVertices; v += (int) blockDim.x) {
            if (v != s && delta[v] != 0.0) {
                centrality[v] += delta[v] * (double) sigma[v] - 1;
            }
        }
    }
}

Graph *copyGraphToCuda(Graph *graph, vector<int*> &devGraphArrays) {
    Graph *deviceGraph;

    // Allocate and copy graph object to cuda
    cudaCheck(hipMalloc((void **) &deviceGraph, sizeof(Graph)));
    cudaCheck(hipMemcpy(deviceGraph, graph, sizeof(Graph), hipMemcpyHostToDevice));

    // Copy arrays that graph stores pointers to
    int *vmap, *vptrs, *adjs;
    cudaCheck(hipMalloc((void **) &vmap, sizeof(int) * graph->vmapSize));
    cudaCheck(hipMemcpy(vmap, graph->vmap, sizeof(int) * graph->vmapSize, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(&(deviceGraph->vmap), &vmap, sizeof(int *), hipMemcpyHostToDevice));

    cudaCheck(hipMalloc((void **) &vptrs, sizeof(int) * graph->vptrsSize));
    cudaCheck(hipMemcpy(vptrs, graph->vptrs, sizeof(int) * graph->vptrsSize, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(&(deviceGraph->vptrs), &vptrs, sizeof(int *), hipMemcpyHostToDevice));

    cudaCheck(hipMalloc((void **) &adjs, sizeof(int) * graph->adjsSize));
    cudaCheck(hipMemcpy(adjs, graph->adjs, sizeof(int) * graph->adjsSize, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(&(deviceGraph->adjs), &adjs, sizeof(int *), hipMemcpyHostToDevice));

    devGraphArrays = {vmap, vptrs, adjs};

    return deviceGraph;
}

void freeGraphFromCuda(Graph *deviceGraph, vector<int *> &devGraphArrays) {
    for (auto arrPtr: devGraphArrays) {
        cudaCheck(hipFree(arrPtr));
    }
    cudaCheck(hipFree(deviceGraph));
}

__constant__ int NUM_THREADS = BLOCKS * THREADS;
__device__ bool devCont;

__global__ void initArrays(Graph *graph, int *d, int *sigma, int s) {
    int idx = (int) (threadIdx.x + blockIdx.x * blockDim.x);

    for (int v = idx; v < graph->numVertices; v += NUM_THREADS) {
        if (v == s) {
            d[s] = 0;
            sigma[s] = 1;
        }
        else {
            d[v] = -1;
            sigma[v] = 0;
        }
    }
}

__global__ void forwardStep(Graph *graph, int *d, int *sigma, int l) {
    int idx = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int v, w;

    for (int i = idx; i < graph->vmapSize; i += NUM_THREADS) {
        w = graph->vmap[i];

        if (d[w] == l) {
            for (int j = (int) graph->vptrs[i]; j < graph->vptrs[i + 1]; ++j) {
                v = graph->adjs[j];

                if (d[v] == -1) {
                    d[v] = l + 1;  // d[w] + 1
                    devCont = true;
                }

                if (d[v] == l + 1) {  // (d[v] == d[w] + 1)
                    atomicAdd(&sigma[v], sigma[w]);
                }
            }
        }
    }
}

__global__ void initDelta(Graph *graph, int *sigma, double *delta) {
    int idx = (int) (threadIdx.x + blockIdx.x * blockDim.x);

    for (int v = idx; v < graph->numVertices; v += NUM_THREADS) {
        delta[v] = (sigma[v] != 0) ? (1 / (double) sigma[v]) : 0.0;
    }
}

__global__ void backwardStep(Graph *graph, int *d, double *delta, int l) {
    int idx = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int v, w;
    double sum;

    for (int i = idx; i < graph->vmapSize; i += NUM_THREADS) {
        w = graph->vmap[i];

        if (d[w] == l) {
            sum = 0;

            for (int j = (int) graph->vptrs[i]; j < graph->vptrs[i + 1]; ++j) {
                v = graph->adjs[j];

                if (d[v] == l + 1) {  // (d[v] == d[w] + 1)
                    sum += delta[v];
                }
            }
            atomicAdd(&delta[w], sum);
        }
    }
}

__global__ void updateCentrality(Graph *graph, int *sigma, double *centrality, double *delta, int s) {
    int idx = (int) (threadIdx.x + blockIdx.x * blockDim.x);

    for (int v = idx; v < graph->numVertices; v += NUM_THREADS) {
        if (v != s && delta[v] != 0.0) {
            centrality[v] += delta[v] * (double) sigma[v] - 1;
        }
    }
}

void runKernels(Graph *graph, Graph *deviceGraph, int *d, int *sigma, double *deviceCentrality, double *delta) {
    int s = 0, l;
    bool cont;

    while (s < graph->numVertices) {
        l = 0;
        cont = true;

        // Init arrays in parallel
        initArrays<<<BLOCKS, THREADS>>>(deviceGraph, d, sigma, s);

        // Forward pass
        while (cont) {
            cont = false;

            // Forward step in parallel
            cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(devCont), &cont, sizeof(bool)));
            forwardStep<<<BLOCKS, THREADS>>>(deviceGraph, d, sigma, l);
            cudaCheck(hipMemcpyFromSymbol((void *) &cont, devCont, sizeof(bool)));

            ++l;
        }

        initDelta<<<BLOCKS, THREADS>>>(deviceGraph, sigma, delta);

        // Backward pass
        while (l > 1) {
            --l;

            // Backward step in parallel
            backwardStep<<<BLOCKS, THREADS>>>(deviceGraph, d, delta, l);
        }

        // Update centrality values
        updateCentrality<<<BLOCKS, THREADS>>>(deviceGraph, sigma, deviceCentrality, delta, s);
        ++s;
    }
}

double *runBrandesCuda(int numVertices, int numEdges, int **edges) {
    // Pointers to arrays stored in graph that should be freed from cuda
    vector<int *> devGraphArrays;
    // Create graph and copy it to device
    Graph graph = Graph(numVertices, numEdges, edges);
    Graph *deviceGraph = copyGraphToCuda(&graph, devGraphArrays);

    // Create all necessary arrays, allocate them and copy centrality to device
    int *d, *sigma;
    double *centrality = (double *) calloc(graph.numVertices, sizeof(double)),
           *deviceCentrality, *delta;

    cudaCheck(hipMalloc((void **)&sigma, sizeof(int) * graph.numVertices));
    cudaCheck(hipMalloc((void **)&d, sizeof(int) * graph.numVertices));
    cudaCheck(hipMalloc((void **)&deviceCentrality, sizeof(double) * graph.numVertices));
    cudaCheck(hipMalloc((void **)&delta, sizeof(double) * graph.numVertices));
    cudaCheck(hipMemcpy(deviceCentrality, centrality, sizeof(double) * graph.numVertices, hipMemcpyHostToDevice));

    // Timer
    hipEvent_t start, stop;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));
    cudaCheck(hipEventRecord(start, nullptr));

    runKernels(&graph, deviceGraph, d, sigma, deviceCentrality, delta);
    // brandesCudaSingleBlock<<<1, THREADS>>>(deviceGraph, deviceCentrality, delta, d, sigma);

    // Record elapsed time and destroy events
    cudaCheck(hipEventRecord(stop, nullptr));
    cudaCheck(hipEventSynchronize(stop));

    float timeMilis, timeSeconds, timeMinutes, timeMinutesRemSeconds;
    cudaCheck(hipEventElapsedTime(&timeMilis, start, stop));
    timeSeconds = timeMilis / 1000;
    timeMinutesRemSeconds = (float) fmod(timeSeconds, 60);
    timeMinutes = (timeSeconds - timeMinutesRemSeconds) / 60;
    printf("Elapsed time: %3.1f ms | %.1f s | %.0f min %.1f s\n",
           timeMilis, timeSeconds, timeMinutes, timeMinutesRemSeconds);

    cudaCheck(hipEventDestroy(start));
    cudaCheck(hipEventDestroy(stop));

    // Copy centrality and clean memory
    cudaCheck(hipMemcpy(centrality, deviceCentrality, sizeof(double) * graph.numVertices, hipMemcpyDeviceToHost));
    freeGraphFromCuda(deviceGraph, devGraphArrays);
    cudaCheck(hipFree(sigma));
    cudaCheck(hipFree(d));
    cudaCheck(hipFree(deviceCentrality));
    cudaCheck(hipFree(delta));

    hipDeviceReset();  // TODO For cuda-memcheck

    return centrality;
}
